
#include <hip/hip_runtime.h>
//#include <iostream>
//#include <math.h>
//using namespace std;
//// Kernel function to add the elements of two arrays
//__global__
//void add(int n, float *x, float *y)
//{
//    int index = blockIdx.x * blockDim.x + threadIdx.x;
//    int stride = blockDim.x * gridDim.x;
//    for (int i = index; i < n; i += stride)
//        y[i] = x[i] + y[i];
//}
//
//int main(void)
//{
//    int N = 1;
//    N = N << 31;
//    float *x, *y;
//
//    // Allocate Unified Memory – accessible from CPU or GPU
//    cudaMallocManaged(&x, N*sizeof(float));
//    cudaMallocManaged(&y, N*sizeof(float));
//
//    // initialize x and y arrays on the host
//    for (int i = 0; i < N; i++) {
//        x[i] = 0.00000001f;
//        y[i] = 0.00000003f;
//    }
//
//    for(int temp = 0; temp<1000000000; temp++)
//    {
////        cout << temp;
//        // Run kernel on 1M elements on the GPU
//
//        int blockSize = 4096*8;
//        int numBlocks = (N + blockSize - 1) / blockSize;
//        add<<<numBlocks, blockSize>>>(N, x, y);
//
//        // Wait for GPU to finish before accessing on host
//        cudaDeviceSynchronize();
//        if (temp% 100000 == 9999){
//            cout << temp << "\n";
//            cout << y[0] << "\n";
//        }
//    }
//
//
//    // Check for errors (all values should be 3.0f)
//    float maxError = 0.0f;
//    for (int i = 0; i < N; i++)
//        maxError = fmax(maxError, fabs(y[i]-3.0f));
//    std::cout << "Max error: " << maxError << std::endl;
//
//    // Free memory
//    cudaFree(x);
//    cudaFree(y);
//
//    return 0;
//}